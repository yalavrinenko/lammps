#include "hip/hip_runtime.h"
#include "../DataTypes.hpp"
#include "../utils/pls.hpp"
#include "../xcfunctionals/LDA.hpp"
#include "../xcfunctionals/LSDA.hpp"
#include "../xcfunctionals/ModLDA.hpp"
#include "CudaKernels.hpp"
#include "awpmd-dft-nvgpu.hpp"
#include "cuApproxTableObject.hpp"
#include "gpu_api.h"
#include "gpu_functors.hpp"
#include "gpu_integration.hpp"
#include "numeric_gpu.cuh"
#include <chrono>
#include <iostream>

bool XCEnergy_nvgpu::check_texture_object_support(){
  hipDeviceProp_t cuda_dev_prop{};
  int current_device;
  hipGetDevice(&current_device);
  hipGetDeviceProperties(&cuda_dev_prop, current_device);

  return m_config.use_xc_tables && cuda_dev_prop.major >= 3;
}

class XCApproximation {
public:
  static std::unique_ptr<gpuApproxTablesObject> &gpu_approximation_table() {
    static std::unique_ptr<gpuApproxTablesObject> gpu_approximation_table_ =
        nullptr;
    return gpu_approximation_table_;
  }
};

XCEnergy::XCResult
XCEnergy_nvgpu::energy(std::vector<class WavePacket> const &wp_spin_up,
                       std::vector<class WavePacket> const &wp_spin_down,
                       bool calc_force) {
  if (m_config.approximation->Type == ApproxType::T_VOID) {
    return XCResult{0.0f, 0.0f, {}};
  }

  auto total_packet_count = wp_spin_down.size() + wp_spin_up.size();
  m_packet_count = total_packet_count;

  host_allocation(m_packet_count);
  std::vector<WavePacketInfo> wp;
  std::transform(wp_spin_up.begin(), wp_spin_up.end(), std::back_inserter(wp),
                 [calc_force](WavePacket const &p) {
                   return WavePacketInfo{p.get_r().get_ptr(), p.get_width(), ElectronSpin::E_UP, calc_force};
                 });
  std::transform(wp_spin_down.begin(), wp_spin_down.end(),
                 std::back_inserter(wp), [calc_force](WavePacket const &p) {
        return WavePacketInfo{p.get_r().get_ptr(), p.get_width(), ElectronSpin::E_DOWN, calc_force};
      });
  copy_packet_to_host(wp);

  XCEnergy::Energy xcenergy{};
  std::vector<std::vector<float>> derivatives;
  if (!m_config.use_adaptive_mesh) {
    std::tie(xcenergy.potential, xcenergy.kinetic) =
        regular_mesh_integration(derivatives, calc_force);
  } else {
    std::tie(xcenergy.potential, xcenergy.kinetic) =
        adaptive_mesh_integration(derivatives, calc_force, wp);
  }

  xcenergy.potential *= units().Hartree2Energy;
  xcenergy.kinetic *= units().Hartree2Energy;

  for (auto &va : derivatives)
    for (auto &a : va) {
      a *= units().Hartree2Energy * units().Distance2Bohr;
    }

  return XCResult(xcenergy.potential, xcenergy.kinetic, std::move(derivatives));
}

void XCEnergy_nvgpu::init_xc_approximation() {
  Logger::Info("Init XCApprox on GPU");
  IApproximation **ptr_ref;

  if (m_config.approximation_device != nullptr) {
    ::__cudft_remove_xc_approximation<<<1, 1>>>(m_config.approximation_device);
    m_config.approximation_device = nullptr;
  }

  SAFECALL(hipMalloc(&ptr_ref, sizeof(IApproximation *)),
           "Error in hipMalloc for xc_approx");

  switch (m_config.approximation->Type) {
    case ApproxType::T_LDA:
      ::__cudft_init_xc_approximation<LDA>
      <<<1, 1>>>(ptr_ref, 0.738558766f, -0.01554534543482745f, 20.4562557f);
      break;
    case ApproxType::T_LDA_2:
      ::__cudft_init_xc_approximation<ModLDA><<<1, 1>>>(ptr_ref);
      break;
    case ApproxType::T_LSDA:
      ::__cudft_init_xc_approximation<LSDA><<<1, 1>>>(ptr_ref);
      break;
    case ApproxType::T_VOID:
      ::__cudft_init_xc_approximation<VoidApproximation><<<1, 1>>>(ptr_ref);
    case ApproxType::T_DUMMY:
      ::__cudft_init_xc_approximation<DummyApproximation><<<1, 1>>>(ptr_ref);
      break;
  }
  SAFECALL(hipDeviceSynchronize(), "Error in thread sync for xc_approx");

  IApproximation **ptr = new IApproximation *[1];
  SAFECALL(hipMemcpy(ptr, ptr_ref, sizeof(IApproximation *),
                      hipMemcpyDeviceToHost),
           "Error in hipMemcpy for xc_approx");

  Logger::Info("XCApprox GPU Ptr:", *ptr);
  m_config.approximation_device = *ptr;

//#ifndef CUDA_NO_TEXTURE_OBJ
  if (check_texture_object_support()) {
    if (m_config.approximation->Type == ApproxType::T_LSDA) {
      Logger::Info("Texture object is available. Switch to table xc-functional.");

      auto approx_table_props = ApproximationTableProps();
      approx_table_props.density_range[0] = static_cast<float>(m_config.packet_number) * 0.0001f;
      approx_table_props.density_range[1] = static_cast<float>(m_config.packet_number);

      XCApproximation::gpu_approximation_table() =
          std::unique_ptr<gpuApproxTablesObject>(new gpuApproxTablesObject(
              m_config.approximation, m_config.approximation_device,
              approx_table_props));

      ::__cudft_init_xc_approximation<TableApproximation<gpuApproxTablesObject>>
      <<<1, 1>>>(ptr_ref,
                 *(XCApproximation::gpu_approximation_table().get()));
      SAFECALL(hipDeviceSynchronize(), "Error in thread sync for xc_approx");
      SAFECALL(hipMemcpy(ptr, ptr_ref, sizeof(IApproximation *), hipMemcpyDeviceToHost),
               "Error in hipMemcpy for xc_approx");
      Logger::Info("New XCApprox GPU Ptr:", *ptr);
      m_config.approximation_device = *ptr;
    }
  }
//#endif

  Logger::Info("Check...");
  ::__cudft_xc_check<<<1, 8>>>(m_config.approximation_device);
  SAFECALL(hipDeviceSynchronize(), "Error in thread sync for check xc_approx");

  Logger::Info("Done.");
}

XCEnergy_nvgpu::XCEnergy_nvgpu(unsigned int packet_number,
                               DFTConfig const &meshConfig, long gpu_id) {
  Logger::ModuleName("AWPMD-DFT GPU");

  Logger::Warning("AWPMD-DFT VERSION FROM", __DATE__);

  hipSetDevice(gpu_id);

  m_packet_count = packet_number;
  Logger::Info("Init dft module on GPU =", gpu_id,
               " Packet number =", m_packet_count);

  setConfig(meshConfig);

  Logger::Info("Allocate",
               m_packet_count * sizeof(GaussPacket<float>) / (1024.0 * 1024.0),
               "Mb in host memory for packet.");
  host_allocation(m_packet_count);

  Logger::Info("Allocate",
               m_packet_count * sizeof(GaussPacket<float>) / (1024.0 * 1024.0),
               "Mb in device memory for packet.");
  device_allocation(m_packet_count);
}

XCEnergy_nvgpu::~XCEnergy_nvgpu() {
  if (XCApproximation::gpu_approximation_table() != nullptr)
    XCApproximation::gpu_approximation_table()->free();
}

void XCEnergy_nvgpu::device_allocation(size_t new_size) {

  if (!m_config.use_adaptive_mesh) {
    if (raw_energy.count() == 0) {
      raw_energy = Mesh_nvgpu(m_config.mesh_size.size.as_struct.x,
                              m_config.mesh_size.size.as_struct.y,
                              m_config.mesh_size.size.as_struct.z);

      raw_kinetic = Mesh_nvgpu(m_config.mesh_size.size.as_struct.x,
                               m_config.mesh_size.size.as_struct.y,
                               m_config.mesh_size.size.as_struct.z);

      rho = ElectronDensity<Mesh_nvgpu>(
          (m_config.mesh_size.size.as_struct.x + 1) * 2,
          m_config.mesh_size.size.as_struct.y * 2 + 2,
          m_config.mesh_size.size.as_struct.z * 2 + 2);
    }

    if (m_config.calc_force && raw_derivatives.m_depth < new_size) {
      raw_derivatives = Mesh_nvgpu(PacketType::derivatives_count(),
                                   m_config.mesh_size.size.as_struct.x *
                                   m_config.mesh_size.size.as_struct.y *
                                   m_config.mesh_size.size.as_struct.z,
                                   new_size);
    }
  }

  if (dev_mem_size < new_size) {
    SAFECALL(hipFree(device_mem), "Error in memory reallocation.");
    SAFECALL(hipMalloc(&device_mem, new_size * sizeof(GaussPacket<float>)),
             "Error in allocation memory for packet");
    dev_mem_size = new_size;
  }
}

std::tuple<double, double> XCEnergy_nvgpu::adaptive_mesh_integration(std::vector<std::vector<float>> &derivatives,
    bool calc_force, const vector<WavePacketInfo> &wavepackets) {

  Integrator_nvgpu<MeshCell> energy_integration_engine_;

  integrator_.refine_mesh(
      MeshCell::RangeType(m_config.mesh_start.size.as_struct), MeshCell::RangeType(m_config.mesh_fin.size.as_struct),
      [this](MeshCell::RangeType a, MeshCell::RangeType b) {
        return this->refine_mesh_condition(host_mem.begin(), host_mem.end(), a, b);
      },
      MeshhWorldTopo{m_config.nodes, m_config.node_rank});

  energy_functor_sic<MeshCell, PacketType> energy_eval{};
  energy_eval.approx = m_config.approximation_device;

  auto xck_energy = integrator_.integrate<NumericType<float, float>>(
      energy_integration_engine_, host_mem, energy_eval);

  if (calc_force) {
    Integrator_nvgpu<ForceMeshCell> force_integration_engine_;

    force_integrator_.clear();
    auto nbins = m_config.force_cell_bins;

    for (auto packet_index = 0; packet_index < host_mem.size();
         ++packet_index) {
      if (wavepackets[packet_index].calc_force) {
        auto const &packet = host_mem[packet_index];

        float volume_dh =
            packet.width * static_cast<PacketType::fields_type>(m_config.max_distance);

        ForceMeshCell::RangeType pbegin{packet.r[0] - volume_dh,
                                        packet.r[1] - volume_dh,
                                        packet.r[2] - volume_dh};
        ForceMeshCell::RangeType pend{packet.r[0] + volume_dh,
                                      packet.r[1] + volume_dh,
                                      packet.r[2] + volume_dh};

        force_integrator_.refine_linked_mesh(pbegin, pend, nbins, packet_index);
      }
    }

    derivatives = std::move(
        std::vector<std::vector<float>>(m_packet_count, std::vector<float>(PacketType::derivatives_count(), 0)));

    auto derivative_eval = derivatives_functor_sic<ForceMeshCell, PacketType>::from_energy_functor(energy_eval);

    auto batch_size = nbins * nbins * nbins;
    auto forces =
        force_integrator_.integrate<std::vector<NumericType<float, float, float, float>>>(
                force_integration_engine_, host_mem, batch_size, derivative_eval);

    auto force_it = forces.begin();
    for (auto packet_index = 0u; packet_index < m_packet_count;
         ++packet_index) {
      float dw = 0.0f, dx = 0.0f, dy = 0.0f, dz = 0.0f;
      if (wavepackets[packet_index].calc_force) {
        thrust::tie(dx, dy, dz, dw) = *force_it;
        ++force_it;
      }

      auto round_float = [](float v) {
        return std::abs(static_cast<double>(v)) <= std::numeric_limits<float>::epsilon() ? 0.0 : static_cast<double>(v);
      };

      derivatives[packet_index][0] += round_float(dx);
      derivatives[packet_index][1] += round_float(dy);
      derivatives[packet_index][2] += round_float(dz);
      derivatives[packet_index][3] += round_float(dw);
    };
  }

  return std::make_tuple<double, double>(thrust::get<0>(xck_energy),
                                         thrust::get<1>(xck_energy));
}

XCEnergy::XCResult
XCEnergy_nvgpu::energy(const vector<WavePacketInfo> &wavepackets,
                       bool calc_force) {
  if (m_config.approximation->Type == ApproxType::T_VOID) {
    return XCResult{0.0f, 0.0f, {}};
  }

  m_packet_count = wavepackets.size();

  host_allocation(m_packet_count);
  copy_packet_to_host(wavepackets);

  XCEnergy::Energy xcenergy{};
  std::vector<std::vector<float>> derivatives;
  if (!m_config.use_adaptive_mesh) {
    std::tie(xcenergy.potential, xcenergy.kinetic) =
        regular_mesh_integration(derivatives, calc_force);
  } else {
    std::tie(xcenergy.potential, xcenergy.kinetic) =
        adaptive_mesh_integration(derivatives, calc_force, wavepackets);
  }

  xcenergy.potential *= units().Hartree2Energy;
  xcenergy.kinetic *= units().Hartree2Energy;

  for (auto &va : derivatives)
    for (auto &a : va) {
      a *= units().Hartree2Energy * units().Distance2Bohr;
    }

  return XCResult(xcenergy.potential, xcenergy.kinetic, std::move(derivatives));
}

// NOT IMPL YET!
std::tuple<double, double> XCEnergy_nvgpu::regular_mesh_integration(
    std::vector<std::vector<float>> &derivatives, bool ) {
  throw std::string("Regular mesh not supported!");
  /*device_allocation(m_packet_count);

  thrust::copy(host_mem.begin(), host_mem.end(),
  thrust::device_ptr<PacketType>(device_mem));

  compute_density(device_mem);
  auto xcenergy = this->compute_energy(device_mem);

  derivatives = std::move(((calc_force) ? this->compute_derivatives(device_mem)
  : std::vector<std::vector<float>>{})); return std::tuple<double,
  double>(xcenergy.potential, xcenergy.kinetic);*/
}

XCEnergy::Energy XCEnergy_nvgpu::compute_energy(PacketType *) {
  throw std::string("Regular mesh not supported!");
  /*KernelExecutionConfiguration config;
  config.positive = {m_spin_up_index.start, m_spin_up_index.end};

  config.space_delta = {m_config.mesh_step.size.as_struct.x,
  m_config.mesh_step.size.as_struct.y, m_config.mesh_step.size.as_struct.z};
  config.space_shift = {(float) m_config.mesh_start[0], (float)
  m_config.mesh_start[1], (float) m_config.mesh_start[2]};

  std::tie(config.width, config.height, config.depth) =
  std::tie(m_config.mesh_size.size.as_array[0],
                                                                 m_config.mesh_size.size.as_array[1],
                                                                 m_config.mesh_size.size.as_array[2]);
  config.packets_count = m_packet_count;

  dim3 block(8, 8, 4);
  dim3 grid(m_config.mesh_size.size.as_struct.x / block.x,
  m_config.mesh_size.size.as_struct.y / block.y,
            m_config.mesh_size.size.as_struct.z / block.z);

  unsigned long shared_mem_size = block.x * block.y * block.z *
  sizeof(PacketType);

  raw_energy.clear();
  raw_kinetic.clear();

  auto CuError = launch_kernel_eng(KernelConfig{grid, block, shared_mem_size},
  m_config.approximation_device, rho, raw_energy, raw_kinetic, config);

  if (CuError != hipSuccess) {
    Logger::Error("Error in start cuda kernels. [", __FUNCTION__, __LINE__, "]",
  hipGetErrorString(CuError)); std::terminate();
  }

  SAFECALL(hipDeviceSynchronize(), "Error in thread sync.");

  auto make_reduction = [this](float *ptr) -> float {
    thrust::device_ptr<float> raw_ptr(ptr);
    return thrust::reduce(raw_ptr, raw_ptr + raw_energy.count(), 0.0f,
  thrust::plus<float>());
  };

  auto potential_energy = make_reduction(raw_energy.dev_ptr());
  auto kinetic_energy = make_reduction(raw_kinetic.dev_ptr());

  return {potential_energy, kinetic_energy};*/
}

std::vector<std::vector<float>>
XCEnergy_nvgpu::compute_derivatives(PacketType const *) {
  throw std::string("Regular mesh not supported!");
  /*KernelExecutionConfigurationDerivs config{};

  config.space_delta = {m_config.mesh_step.size.as_struct.x,
                        m_config.mesh_step.size.as_struct.y,
                        m_config.mesh_step.size.as_struct.z};

  config.space_shift = {(float) m_config.mesh_start[0],
                        (float) m_config.mesh_start[1],
                        (float) m_config.mesh_start[2]};

  std::tie(config.width, config.height, config.depth) =
  std::tie(m_config.mesh_size.size.as_array[0],
                                                                 m_config.mesh_size.size.as_array[1],
                                                                 m_config.mesh_size.size.as_array[2]);
  config.packets_count = m_packet_count;

  dim3 block(8, 8, 4);
  dim3 grid(m_config.mesh_size.size.as_struct.x / block.x,
  m_config.mesh_size.size.as_struct.y / block.y,
            m_config.mesh_size.size.as_struct.z / block.z);

  unsigned long shared_mem_size = block.x * block.y * block.z *
  sizeof(PacketType);

  raw_derivatives.clear();
  auto cuError = launch_kernel_derivs({grid, block, shared_mem_size},
  m_config.approximation_device, rho, packets, raw_derivatives, config);

  if (cuError != hipSuccess) {
    Logger::Error("Error in start cuda kernels. [", __FUNCTION__, __LINE__, "]",
  hipGetErrorString(cuError)); std::terminate();
  }

  SAFECALL(hipDeviceSynchronize(), "Error in thread sync.");

  auto make_reduction = [this](float *begin, size_t count) -> float {
    thrust::device_ptr<float> raw_begin(begin);
    thrust::device_ptr<float> raw_end(begin + count);
    return thrust::reduce(raw_begin, raw_end, 0.0f, thrust::plus<float>());
  };

  std::vector<std::vector<float>> derivatives(m_packet_count,
  std::vector<float>(PacketType::derivatives_count(), 0));

  for (auto i = 0u; i < m_packet_count; ++i) {
    auto space_base = raw_derivatives.m_height;
    auto particle_shift = space_base * raw_derivatives.m_width;
    auto ptr = raw_derivatives.dev_ptr() + i * particle_shift;
    for (auto k = 0u; k < raw_derivatives.m_width; ++k) {
      derivatives[i][k] = make_reduction(ptr + k * space_base, space_base);
    }
  }

  return derivatives;*/
}

void XCEnergy_nvgpu::compute_density(XCEnergy::PacketType *) {
  throw std::string("Regular mesh not supported!");
  /*KernelExecutionConfiguration config{};
  config.positive = {m_spin_up_index.start, m_spin_up_index.end};

  config.space_delta = {m_config.mesh_step.size.as_struct.x / 2.0f,
  m_config.mesh_step.size.as_struct.y / 2.0f,
                        m_config.mesh_step.size.as_struct.z / 2.0f};
  config.space_shift = {
      (float) m_config.mesh_start[0],
      (float) m_config.mesh_start[1],
      (float) m_config.mesh_start[2]
  };

  std::tie(config.width, config.height, config.depth) =
  std::tie(rho.total.m_width, rho.total.m_height, rho.total.m_depth);
  config.packets_count = m_packet_count;

  dim3 block(8, 8, 4);
  dim3 grid(config.width / block.x + 1, config.height / block.y + 1,
            config.depth / block.z + 1);

  unsigned long shared_mem_size = block.x * block.y * block.z *
  sizeof(PacketType);
  __CuDftDensityCalculation << < grid, block, shared_mem_size >> > (packets,
  rho, config); auto CuError = hipGetLastError();

  if (CuError != hipSuccess) {
    Logger::Error("Error in start cuda kernels. [", __FUNCTION__, __LINE__, "]",
  hipGetErrorString(CuError)); std::terminate();
  }

  SAFECALL(hipDeviceSynchronize(), "Error in thread sync.");*/
}